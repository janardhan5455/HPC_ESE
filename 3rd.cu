#include "hip/hip_runtime.h"
%%cu
#include<stdio.h>
#include<hip/hip_runtime.h>
__global__ void matadd(int *l,int *m, int *n)
{
    int x=blockIdx.x;
    int y=blockIdx.y;
    int id=gridDim.x * y +x;
    n[id]=l[id]+*m;
}
int main()
{
    int a[2][3];
    int b=5;
    int c[2][3];
    int *d,*e,*f;
    int i,j;
    
    for(i=0;i<2;i++)
    {
        for(j=0;j<3;j++)
            {
                a[i][j]=i;
            }
    }
    
    
    hipMalloc((void **)&d,2*3*sizeof(int));
    hipMalloc((void **)&e,sizeof(int));
    hipMalloc((void **)&f,2*3*sizeof(int));
    hipMemcpy(d,a,2*3*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(e,&b,sizeof(int),hipMemcpyHostToDevice);
    
    dim3 grid(3,2);


    matadd<<<grid,1>>>(d,e,f);

    hipMemcpy(c,f,2*3*sizeof(int),hipMemcpyDeviceToHost);
    printf("\nSum of matrix and scalar:\n ");
    for(i=0;i<2;i++)
    {
        for(j=0;j<3;j++)
        {
              printf("%d\t",c[i][j]);
        }
        printf("\n");
    }
    hipFree(d);
    hipFree(e);
    hipFree(f);
    return 0;
}